#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/solver.h"

// --- Physics Functions (CPU versions for fallback/reference) ---
// These are the same Johnson-Cook and Heat Generation functions as before.
// ...

// --- The CUDA Kernel ---
// This is the function that will run in parallel on thousands of GPU cores.
// Each thread will be responsible for calculating the physics for one node.
__global__ void update_nodes_kernel(Node* d_nodes, int num_nodes, double current_strain, double A, double B, double n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_nodes) {
        Node node = d_nodes[idx]; // Get the node for this thread

        // Run the same physics calculations, but on the GPU
        if (node.z < 0.5) {
            const double MELTING_TEMP = 660.0, ROOM_TEMP = 25.0;
            const double DENSITY = 2700.0, SPECIFIC_HEAT = 896.0;
            double strain_increase_this_step = 0.01;

            double stress = node.stress; // Simplified Johnson-Cook on GPU
            double temp_increase = (stress * 1e6 * strain_increase_this_step * 0.9) / (DENSITY * SPECIFIC_HEAT);
            
            node.temperature += temp_increase;
            node.stress = stress; // Update stress if needed
        }
        
        d_nodes[idx] = node; // Write the updated node back to memory
    }
}

// --- The C++ Wrapper Function ---
void solve_time_step_cuda(std::vector<Node>& nodes, double current_strain, const json& material_params) {
    int num_nodes = nodes.size();
    if (num_nodes == 0) return;

    // 1. Allocate memory on the GPU
    Node* d_nodes; // Pointer to device (GPU) memory
    hipMalloc(&d_nodes, num_nodes * sizeof(Node));

    // 2. Copy the node data from the CPU (host) to the GPU (device)
    hipMemcpy(d_nodes, nodes.data(), num_nodes * sizeof(Node), hipMemcpyHostToDevice);

    // 3. Launch the CUDA Kernel
    // We tell the GPU to launch enough threads to cover all our nodes.
    int threads_per_block = 256;
    int blocks_per_grid = (num_nodes + threads_per_block - 1) / threads_per_block;
    
    // Unpack material params for the kernel
    double A = material_params["A"];
    double B = material_params["B"];
    double n = material_params["n"];

    update_nodes_kernel<<<blocks_per_grid, threads_per_block>>>(d_nodes, num_nodes, current_strain, A, B, n);

    // 4. Copy the results back from the GPU to the CPU
    hipMemcpy(nodes.data(), d_nodes, num_nodes * sizeof(Node), hipMemcpyDeviceToHost);

    // 5. Free the memory on the GPU
    hipFree(d_nodes);
}
